#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <torch/types.h>
#include <vector>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])

// FP32
// Swish x: N, y: N y=x*sigmoid(x)
__device__ __forceinline__ float swish(float x) {
  return x / (1.0f + expf(-x));
}

__global__ void swish_f32_kernel(float *x, float *y, int N) {}

__global__ void swish_f32x4_kernel(float *x, float *y, int N) {}

//  FP16
__device__ __forceinline__ half swish_half(half x) {}

__global__ void swish_f16_kernel(half *x, half *y, int N) {}

__global__ void swish_f16x2_kernel(half *x, half *y, int N) {}

__global__ void swish_f16x8_kernel(half *x, half *y, int N) {}

__global__ void swish_f16x8_pack_kernel(half *x, half *y, int N) {}

#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }

#define TORCH_BINDING_SWISH(packed_type, th_type, element_type, n_elements)    \
  void swish_##packed_type(torch::Tensor x, torch::Tensor y) {                 \
    CHECK_TORCH_TENSOR_DTYPE(x, (th_type))                                     \
    CHECK_TORCH_TENSOR_DTYPE(y, (th_type))                                     \
    const int ndim = x.dim();                                                  \
    if (ndim != 2) {                                                           \
      int N = 1;                                                               \
      for (int i = 0; i < ndim; ++i) {                                         \
        N *= x.size(i);                                                        \
      }                                                                        \
      dim3 block(256 / (n_elements));                                          \
      dim3 grid((N + 256 - 1) / 256);                                          \
      swish_##packed_type##_kernel<<<grid, block>>>(                           \
          reinterpret_cast<element_type *>(x.data_ptr()),                      \
          reinterpret_cast<element_type *>(y.data_ptr()), N);                  \
    } else {                                                                   \
      const int S = x.size(0);                                                 \
      const int K = x.size(1);                                                 \
      const int N = S * K;                                                     \
      if ((K / (n_elements)) <= 1024) {                                        \
        dim3 block(K / (n_elements));                                          \
        dim3 grid(S);                                                          \
        swish_##packed_type##_kernel<<<grid, block>>>(                         \
            reinterpret_cast<element_type *>(x.data_ptr()),                    \
            reinterpret_cast<element_type *>(y.data_ptr()), N);                \
      } else {                                                                 \
        int N = 1;                                                             \
        for (int i = 0; i < ndim; ++i) {                                       \
          N *= x.size(i);                                                      \
        }                                                                      \
        dim3 block(256 / (n_elements));                                        \
        dim3 grid((N + 256 - 1) / 256);                                        \
        swish_##packed_type##_kernel<<<grid, block>>>(                         \
            reinterpret_cast<element_type *>(x.data_ptr()),                    \
            reinterpret_cast<element_type *>(y.data_ptr()), N);                \
      }                                                                        \
    }                                                                          \
  }

TORCH_BINDING_SWISH(f32, torch::kFloat32, float, 1)
TORCH_BINDING_SWISH(f32x4, torch::kFloat32, float, 4)
TORCH_BINDING_SWISH(f16, torch::kHalf, half, 1)
TORCH_BINDING_SWISH(f16x2, torch::kHalf, half, 2)
TORCH_BINDING_SWISH(f16x8, torch::kHalf, half, 8)
TORCH_BINDING_SWISH(f16x8_pack, torch::kHalf, half, 8)

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(swish_f32)
  TORCH_BINDING_COMMON_EXTENSION(swish_f32x4)
  TORCH_BINDING_COMMON_EXTENSION(swish_f16)
  TORCH_BINDING_COMMON_EXTENSION(swish_f16x2)
  TORCH_BINDING_COMMON_EXTENSION(swish_f16x8)
  TORCH_BINDING_COMMON_EXTENSION(swish_f16x8_pack)
}
